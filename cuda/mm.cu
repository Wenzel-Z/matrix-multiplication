#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <random>
#include <omp.h>
#include <cstdio>

using namespace std;

__global__ void mul_vec_gpu(double* a, double* b, double* c, int size)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < size)
		c[index] = a[index] * b[index];
}

int main(int argc, char** argv)
{
    int N;
    int block_size = 128;
    bool exit = false;
    const int seed = 1;
    if (argc == 1) {
        cout << "Please enter either one or two arguments.\n";
        exit = true;
    }
    else if (argc == 2) {
        N = std::stoi(argv[1]);
    } 
    if (!exit)
    {
        size_t NO_BYTES = N * sizeof(double); //bytes needed
        double *h_a, *h_b, *gpu_result; // host pointers

        // Allocate
        h_a = (double *)malloc(NO_BYTES);
        h_b = (double *)malloc(NO_BYTES);
        gpu_result = (double *)malloc(NO_BYTES);

        // Initialize vectors
        std::mt19937 engine(seed);
        std::uniform_int_distribution<double> dist(-5, 5);

        for (size_t i = 0; i < N; i++) 
        {
            #ifdef TEST
                h_a[i] = 1.0;
            #else
                h_a[i] = dist(engine);
            #endif
        }

        for (size_t i = 0; i < N; i++) 
        {
            #ifdef TEST
                h_b[i] = 1.0;
            #else
                h_b[i] = dist(engine);
            #endif
        }

        memset(gpu_result, 0, NO_BYTES)

        // Cuda Pointers
        double *d_a, *d_b, *d_c;
        hipMalloc((double **)&d_a, NO_BYTES);
        hipMalloc((double **)&d_b, NO_BYTES);
        hipMalloc((double **)&d_c, NO_BYTES);

        
        // Copy from host to device
        hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
	    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

        dim3 block(N);
        dim3 grid((N/ block.x) + 1);

        double gpu_start = omp.get_time();
        mul_vec_gpu <<<grid, block>>> (d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        double gpu_end = omp.get_time();
        total_time = gpu_end - gpu_start;
        printf(total_time);

        double total = 0;
        for (int i = 0; i < N; i++) {
            total = total + c[i];
        }

        printf("%li", total);

        hipFree(d_c);
        hipFree(d_b);
        hipFree(d_a);

        free(gpu_result);
        free(h_a);
        free(h_b);

        hipDeviceReset();
    }
    return 0;
}
